#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void matmul_kernel(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < K) {
        float sum = 0.0f;
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = sum;
    }
}

__global__ void update_kernel(float* X, float* A, float* B, int size, float a, float b, float c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        X[idx] = a * X[idx] + b * A[idx] + c * B[idx];
    }
}

torch::Tensor newtonschulz5_cuda(torch::Tensor G, int steps = 5, float eps = 1e-7) {
    int M = G.size(0);
    int N = G.size(1);
    torch::Tensor X = G.clone().to(torch::kCUDA);
    X /= (X.norm() + eps);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    torch::Tensor A = torch::zeros_like(X);
    torch::Tensor B = torch::zeros_like(X);
    float a = 3.4445f, b = -4.7750f, c = 2.0315f;

    for (int i = 0; i < steps; i++) {
        matmul_kernel<<<numBlocks, threadsPerBlock>>>(X.data_ptr<float>(), X.data_ptr<float>(), A.data_ptr<float>(), M, N, N);
        matmul_kernel<<<numBlocks, threadsPerBlock>>>(A.data_ptr<float>(), A.data_ptr<float>(), B.data_ptr<float>(), M, N, N);
        update_kernel<<<(M * N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(X.data_ptr<float>(), A.data_ptr<float>(), B.data_ptr<float>(), M * N, a, b, c);
    }
    return X;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("newtonschulz5_cuda", &newtonschulz5_cuda, "Muon Optimizer Newton-Schulz CUDA");
}
